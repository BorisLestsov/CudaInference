#include "hip/hip_runtime.h"
#include <stdexcept>
#include "ReluLayer.hpp"
#include "Tensor.hpp"

#include <hipblas.h>
#include <hipblas.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>

#include "helper_functions.cuh"
#include "compute_util.cuh"
#include "npy.hpp"


ReluLayer::ReluLayer(int batch_size_p)
{
    batch_size = batch_size_p;
}

ReluLayer::~ReluLayer(){
    delete _res; 
}

class elwise_max_functor {
    public:
        elwise_max_functor() {}
        __host__ __device__ float operator()(float x) const 
        {
            return fmaxf(x, 0.0);
        }
};

__global__ void relu_ker(float* src, float* dst, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N){
        return;
    }
    dst[i] = fmaxf(0.0, src[i]);
}

void ReluLayer::forward() 
{
    // thrust::device_ptr<float> thr_ptr = thrust::device_pointer_cast<float>(_input->_ptr);
    // thrust::device_ptr<float> thr_ptr2 = thrust::device_pointer_cast<float>(_res->_ptr);
    // thrust::transform(thr_ptr, thr_ptr + _input->count(), thr_ptr2, elwise_max_functor());

    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;
    int N = _input->count();

    num_blocks_x = N/cell_size + (N % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);

    relu_ker<<<grid_size, block_size>>>(_input->_ptr, _res->_ptr, N);
}


void ReluLayer::set_input(Tensor<float>* input)
{
    if (input->size()[0] != batch_size) {
        throw std::runtime_error("batch size does not match");
    }
    _input = input;
    _res = new Tensor<float>(_input->size());
}

Tensor<float>* ReluLayer::get_output()
{
    return _res;
}
