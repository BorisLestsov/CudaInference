#include "LinearLayer.hpp"

#include <hipblas.h>
#include <hipblas.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>

#include "helper_functions.cuh"
#include "compute_util.cuh"


LinearLayer::LinearLayer()
{
    int size;
    size = 5*4;
    hipMalloc(&_w, size*sizeof(float));
    thrust::device_ptr<float> thr_ptr = thrust::device_pointer_cast<float>(_w);
    thrust::fill(thr_ptr, thr_ptr + size, 0.5f);

    size = 3*4;
    hipMalloc(&_res, size*sizeof(float));
    thrust::device_ptr<float> thr_ptr2 = thrust::device_pointer_cast<float>(_res);
    thrust::fill(thr_ptr2, thr_ptr2 + size, 0.0f);

    size = 3*4;
    hipMalloc(&_tmp, size*sizeof(float));
    thrust::device_ptr<float> thr_ptr3 = thrust::device_pointer_cast<float>(_tmp);
    thrust::fill(thr_ptr3, thr_ptr3 + size, 0.0f);
}

void LinearLayer::forward() {}

void LinearLayer::forward_tmp(hipblasHandle_t& cublas_handle, Tensor<float>* input)
{

    row_major_sgemm(cublas_handle, 3, 4, 5, input->_ptr, _w, _res, _tmp);
    hipDeviceSynchronize();

    for (int i = 0; i < 3*4; ++i)
        debug_ker<<<1,1>>>(_res, i);
        hipDeviceSynchronize();

}
