#include <stdexcept>
#include "LinearLayer.hpp"
#include "Tensor.hpp"

#include <hipblas.h>
#include <hipblas.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>

#include "helper_functions.cuh"
#include "compute_util.cuh"
#include "npy.hpp"


LinearLayer::LinearLayer(hipblasHandle_t& cublas_handle, const std::string& w_path, bool bias_p):
    cublas_handle(cublas_handle),
    _bias(bias_p)
{

    std::vector<unsigned long> shape;
    std::vector<float> data;
    bool is_f;

    npy::LoadArrayFromNumpy(w_path + ".weight.npy", shape, is_f, data);
    output_dim = shape[0];
    input_dim = shape[1];

    _w = std::shared_ptr<Tensor<float>>(new Tensor<float>({output_dim, input_dim}));
    _w->from_cpu(data.data());

    if (_bias){
        npy::LoadArrayFromNumpy(w_path + ".bias.npy", shape, is_f, data_b);
    }
}

LinearLayer::~LinearLayer(){
}

void LinearLayer::forward() 
{
    
    if (_bias){
        row_major_sgemm_add(cublas_handle, batch_size, output_dim, input_dim, _input->_ptr, _w->_ptr, _b->_ptr, _res->_ptr, _tmp->_ptr);
    } else {
        row_major_sgemm(cublas_handle, batch_size, output_dim, input_dim, _input->_ptr, _w->_ptr, _res->_ptr, _tmp->_ptr);
    }
}


void LinearLayer::set_input(std::shared_ptr<Tensor<float>> input)
{
    batch_size = input->size()[0];
    int inp_w = input->count() / batch_size;
    if (inp_w != input_dim) {
        throw std::runtime_error(std::string("input dim is different: ") + std::to_string(inp_w) + " vs " + std::to_string(input_dim));
    }

    if (_bias){
        _b = std::shared_ptr<Tensor<float>>(new Tensor<float>({batch_size, output_dim}));
        float* tmp_ptr = _b->_ptr;
        for (int i = 0; i < batch_size; ++i){
            hipMemcpy(tmp_ptr, data_b.data(), output_dim*sizeof(float), hipMemcpyHostToDevice);
            tmp_ptr += output_dim;
        }
    }

    _input = input;
    _tmp = std::shared_ptr<Tensor<float>>(new Tensor<float>({output_dim, batch_size}));
    _res = std::shared_ptr<Tensor<float>>(new Tensor<float>({batch_size, output_dim}));
}

std::shared_ptr<Tensor<float>> LinearLayer::get_output()
{
    return _res;
}

int LinearLayer::get_output_dim()
{
    return output_dim;
}
