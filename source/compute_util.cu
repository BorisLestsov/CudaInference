#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipblas.h>
#include "helper_functions.cuh"

__global__ void debug_ker(float* ptr, int addr){
    //int i = blockIdx.x*blockDim.x + threadIdx.x;
    printf("%d %f\n", addr, ptr[addr]);
}

void debug_array(float* arr, int N){
    for (int i = 0; i < N; ++i){
        debug_ker<<<1,1>>>(arr, i);
    }
    hipDeviceSynchronize();
}

void row_major_sgemm(hipblasHandle_t& cublas_handle, int m, int n, int k, float* A, float* B, float* C, float* tmp){
    float alpha = 1.0;
    float beta = 0.0;
    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, tmp, m));
    checkCublasErrors(hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, tmp, m, &beta, C, n, C, n));
}

void row_major_sgemm_add(hipblasHandle_t& cublas_handle, int m, int n, int k, float* A, float* B, float*D, float* C, float* tmp){
    float alpha = 1.0;
    float beta = 0.0;
    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, tmp, m));
    beta = 1.0;
    checkCublasErrors(hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, tmp, m, &beta, D, n, C, n));
}
