#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipblas.h>
#include "helper_functions.cuh"

__global__ void debug_ker(float* ptr, int addr){
    //int i = blockIdx.x*blockDim.x + threadIdx.x;
    printf("%d %f\n", addr, ptr[addr]);
}

void debug_array(float* arr, int N){
    for (int i = 0; i < N; ++i){
        debug_ker<<<1,1>>>(arr, i);
    }
    hipDeviceSynchronize();
}

void row_major_sgemm(hipblasHandle_t& cublas_handle, int m, int n, int k, float* A, float* B, float* C, float* tmp){
    float alpha = 1.0;
    float beta = 0.0;
    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, tmp, m));
    checkCublasErrors(hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, tmp, m, &beta, C, n, C, n));
}

void row_major_sgemm_add(hipblasHandle_t& cublas_handle, int m, int n, int k, float* A, float* B, float*D, float* C, float* tmp){
    float alpha = 1.0;
    float beta = 0.0;
    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, tmp, m));
    beta = 1.0;
    checkCublasErrors(hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, tmp, m, &beta, D, n, C, n));
}


template<typename T>
__global__ void add_ker(T* src1, T* src2, T* dst, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N){
        return;
    }
    dst[i] = src1[i] + src2[i];
}

template<typename T>
void cuda_add(T* src1, T* src2, T* res, int N){
    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;

    num_blocks_x = N/cell_size + (N % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);
    add_ker<T><<<grid_size, block_size>>>(src1, src2, res, N);
}

template<typename T>
__global__ void sub_ker(T* src1, T* src2, T* dst, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N){
        return;
    }
    dst[i] = src1[i] - src2[i];
}

template<typename T>
void cuda_sub(T* src1, T* src2, T* res, int N){
    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;

    num_blocks_x = N/cell_size + (N % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);
    sub_ker<T><<<grid_size, block_size>>>(src1, src2, res, N);
}


template<typename T>
__global__ void mul_ker(T* src1, T* src2, T* dst, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N){
        return;
    }
    dst[i] = src1[i] * src2[i];
}

template<typename T>
void cuda_mul(T* src1, T* src2, T* res, int N){
    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;

    num_blocks_x = N/cell_size + (N % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);
    mul_ker<T><<<grid_size, block_size>>>(src1, src2, res, N);
}

template<typename T>
__global__ void div_ker(T* src1, T* src2, T* dst, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N){
        return;
    }
    dst[i] = src1[i] / src2[i];
}

template<typename T>
void cuda_div(T* src1, T* src2, T* res, int N){
    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;

    num_blocks_x = N/cell_size + (N % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);
    div_ker<T><<<grid_size, block_size>>>(src1, src2, res, N);
}


template<typename T>
__global__ void transpose_ker(T* src_ptr, T* dst_ptr, int* src_dims, int* strides, int* reorder, int* new_strides, int Ndims, int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    //if (i == 0){
    //    printf("%d \n", total);
    //}
    if (i >= N){
        return;
    }

    int new_idx[10];
    int acc = 0;
    for (int k = 0; k < Ndims; ++k) {
        int cur_i = (i - acc) / strides[k];
        acc += cur_i*strides[k];

        new_idx[reorder[k]] = cur_i;
    }

    int new_i = 0;
    for (int k = 0; k < Ndims; ++k) {
        new_i += new_strides[k]*new_idx[k];
    }

    dst_ptr[new_i] = src_ptr[i];
}

template<typename T>
void cuda_transpose(T* src_ptr, T* dst_ptr, int* src_dims, int* strides, int* reorder, int* new_strides, int Ndims, int N){
    int cell_size = 32;
    dim3 block_size;
    dim3 grid_size;
    int num_blocks_x;
    num_blocks_x = (N)/cell_size + ((N) % cell_size != 0);
    block_size = dim3(cell_size);
    grid_size = dim3(num_blocks_x);

    transpose_ker<<<grid_size, block_size>>>(src_ptr, dst_ptr, src_dims, strides, reorder, new_strides, Ndims, N);
}


template void cuda_add<float>(float*, float*, float*, int);
template void cuda_sub<float>(float*, float*, float*, int);
template void cuda_mul<float>(float*, float*, float*, int);
template void cuda_div<float>(float*, float*, float*, int);

template void cuda_add<int>(int*, int*, int*, int);
template void cuda_sub<int>(int*, int*, int*, int);
template void cuda_mul<int>(int*, int*, int*, int);
template void cuda_div<int>(int*, int*, int*, int);

template void cuda_transpose<float>(float*, float*, int*, int*, int*, int*, int, int);
template void cuda_transpose<int>(int*, int*, int*, int*, int*, int*, int, int);
